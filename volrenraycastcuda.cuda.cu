#include "hip/hip_runtime.h"
#include "volrenraycastcuda.cuda.h"
#include <stdio.h>
#include "cutil_math.h"

static const int maxBlockSize2D = 16;
static texture<float, hipTextureType3D, hipReadModeElementType> volTex;
static texture<float4, hipTextureType2D, hipReadModeElementType> tfTex;
static texture<float4, hipTextureType2D, hipReadModeElementType> entryTex;
static texture<float4, hipTextureType2D, hipReadModeElementType> exitTex;

static dim3 getDimBlock2D(int w, int h)
{
    dim3 dimBlock;
    if (w < maxBlockSize2D)
        dimBlock.x = w;
    else
        dimBlock.x = maxBlockSize2D;
    if (h < maxBlockSize2D)
        dimBlock.y = h;
    else
        dimBlock.y = maxBlockSize2D;
    return dimBlock;
}

static dim3 getDimGrid2D(int w, int h)
{
    dim3 dimGrid;
    if (w < maxBlockSize2D)
        dimGrid.x = 1;
    else
        dimGrid.x = int(ceil(float(w) / maxBlockSize2D));
    if (h < maxBlockSize2D)
        dimGrid.y = 1;
    else
        dimGrid.y = int(ceil(float(h) / maxBlockSize2D));
    return dimGrid;
}

__global__ static void castray(int volWidth, int volHeight, int volDepth,
                        int tfWidth, int tfHeight, float stepSize,
                        float scalarMin, float scalarMax,
                        int texWidth, int texHeight, float* outPtr)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= texWidth || y >= texHeight)
        return;

    float3 entry = make_float3(tex2D(entryTex, x + 0.5f, y + 0.5f));
    float3 exit = make_float3(tex2D(exitTex, x + 0.5f, y + 0.5f));
    float3 dir = normalize(exit - entry);
    float maxLength = length(exit - entry);
    float2 scalar = make_float2(0.f, 0.f);
    float4 acc = make_float4(0.f, 0.f, 0.f, 0.f);
    int step = 0;
    for (; step * stepSize < maxLength; ++step)
    {
        float3 spot = entry + dir * (step * stepSize);
        scalar.x = tex3D(volTex, spot.x * volWidth, spot.y * volHeight, spot.z * volDepth);
        scalar.x = clamp(float((scalar.x - scalarMin) / (scalarMax - scalarMin)), 0.f, 1.f);
        float4 spotColor = tex2D(tfTex, scalar.x * tfWidth, scalar.y * tfHeight);
        acc += spotColor * (1.f - acc.w);
        if (acc.w > 0.999f)
            break;
        scalar.y = scalar.x;
    }
    outPtr[3 * (texWidth * y + x) + 0] = acc.x;
    outPtr[3 * (texWidth * y + x) + 1] = acc.y;
    outPtr[3 * (texWidth * y + x) + 2] = acc.z;
}

void cudacast(int volWidth, int volHeight, int volDepth, hipArray* volArr,
              int tfWidth, int tfHeight, float stepSize, hipTextureFilterMode filter, hipArray* tfArr,
              float scalarMin, float scalarMax,
              int texWidth, int texHeight, hipArray *entryArr, hipArray *exitArr, float *outPtr)
{
    hipBindTextureToArray(volTex, volArr);
    volTex.filterMode = hipFilterModeLinear;
    hipBindTextureToArray(tfTex, tfArr);
    tfTex.filterMode = filter;
    hipBindTextureToArray(entryTex, entryArr);
    entryTex.filterMode = hipFilterModeLinear;
    hipBindTextureToArray(exitTex, exitArr);
    exitTex.filterMode = hipFilterModeLinear;

    dim3 dimBlock = getDimBlock2D(texWidth, texHeight);
    dim3 dimGrid = getDimGrid2D(texWidth, texHeight);
    castray<<<dimGrid, dimBlock>>>(volWidth, volHeight, volDepth,
                                   tfWidth, tfHeight, stepSize,
                                   scalarMin, scalarMax,
                                   texWidth, texHeight, outPtr);

    hipUnbindTexture(exitTex);
    hipUnbindTexture(entryTex);
    hipUnbindTexture(tfTex);
    hipUnbindTexture(volTex);
}
